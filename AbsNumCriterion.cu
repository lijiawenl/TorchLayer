
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/AbsNumCriterion.cu"
#else

void THNN_(AbsNumCriterion_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *target,
           THCTensor *output,
           bool sizeAverage)
{
  THCUNN_check_nElement(state, input, target);
  THCUNN_assertSameGPU(state, 2, input, target);

  ptrdiff_t size = THCTensor_(nElement)(state, input);

  input = THCTensor_(newContiguous)(state, input);
  target = THCTensor_(newContiguous)(state, target);
  /*add by JiaWen Li*/

  /*accreal sum = thrust::inner_product(input_data, input_data+size, target_data, (accreal)0, thrust::plus<accreal>(), abs_functor<real, accreal>());*/
  thrust::device_ptr<real> input_data(THCTensor_(data)(state, input));
  thrust::device_ptr<real> target_data(THCTensor_(data)(state, target));
  accreal sum = thrust::inner_product(input_data, input_data+size, target_data, (accreal)0, thrust::plus<accreal>(), absNum_functor<real, accreal>());

  if (sizeAverage)
    sum /= size;

  THCTensor_(free)(state, input);
  THCTensor_(free)(state, target);
  if(sum >=0){
  THCTensor_(set1d)(state, output, 0, ScalarConvert<accreal, real>::to(sum));
}else{
  THCTensor_(set1d)(state, output, 0, ScalarConvert<accreal, real>::to(-sum));
}
}

void THNN_(AbsNumCriterion_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCTensor *target,
           THCTensor *gradInput,
           bool sizeAverage)
{
  THCUNN_check_nElement(state, input, target);
  THCUNN_assertSameGPU(state, 3, input, target, gradInput);

  ptrdiff_t size = THCTensor_(nElement)(state, input);
  real norm = ScalarConvert<double, real>::to(sizeAverage ? 1./size : 1.);
  real norm1 = ScalarConvert<double, real>::to(sizeAverage ? -1./size : -1.);

  input = THCTensor_(newContiguous)(state, input);
  target = THCTensor_(newContiguous)(state, target);
  
  THCTensor_(resizeAs)(state, gradInput, input);

  thrust::device_ptr<real> input_data(THCTensor_(data)(state, input));
  thrust::device_ptr<real> target_data(THCTensor_(data)(state, target));/*fengzhuang chuanru de zhizhen*/
  thrust::device_ptr<real> gradInput_data(THCTensor_(data)(state, gradInput));
  accreal sum = thrust::inner_product(input_data, input_data+size, target_data, (accreal)0, thrust::plus<accreal>(), absNum_functor<real, accreal>());
 
  if(sum >=0){
  /*thrust::transform(input_data, input_data+size, target_data, gradInput_data, abs_updateGradInput_functor<real>(norm));*/
  thrust::fill(gradInput_data,gradInput_data+size,(real) norm);
  }else{
  thrust::fill(gradInput_data,gradInput_data+size,(real) norm1);
  }
  
  THCTensor_(free)(state, input);
  THCTensor_(free)(state, target);
}

#endif

